#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../common/common.h"


void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++)
        if(abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }

    if (match) printf("Arrays match.\n\n");
}


void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned)time(&t));

    for(int i = 0; i < size; i++)
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
}


void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for(int i = 0; i < N; i++)
        C[i] = A[i] + B[i];
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("now thread idx: %d\n", idx);
    if (idx < N) C[idx] = A[idx] + B[idx];
}

int main(int argc, char **argv) {

    printf("%s Starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;

    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    double iStart, iElaps;

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    int iLen = 1024;
    dim3 block (iLen);
    // dim3 block (1);
    dim3 grid ((nElem + block.x - 1) / block.x);

    iStart = cpuSecond();
    sumArraysOnGPU <<<grid, block>>>(d_A, d_B, d_C, nElem);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("sumArraysOnGPU <<<%d, %d>>> Time elapsed %f sec\n", grid.x, block.x, iElaps);

    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}